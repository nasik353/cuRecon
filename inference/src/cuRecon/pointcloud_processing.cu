#include "hip/hip_runtime.h"
#include "cuRecon/pointcloud_processing.h"
#include "cuRecon/types.h"
#include <hip/hip_runtime.h>
#include <>

#define THREADS 256 
#define WARP_SIZE 32

__forceinline__ __device__ int64_t get_example_idx(int64_t idx,
                                                   const int64_t *ptr,
                                                   const int64_t num_examples) {
  for (int64_t i = 0; i < num_examples; i++) {
    if (ptr[i + 1] > idx)
      return i;
  }
  return num_examples - 1;
}

__global__ void downsamplePointCloudKernel(const point *input, point *output,
                                           int input_size,
                                           float downsample_factor) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int output_size = static_cast<int>(input_size / downsample_factor);

  if (idx < output_size) {
    int input_idx = static_cast<int>(idx * downsample_factor);
    output[idx] = input[input_idx];
  }
}

pointcloud downsamplePointCloud(const pointcloud &input,
                                float downsample_factor) {
  int input_size = input.data.size();
  int output_size = static_cast<int>(input_size / downsample_factor);

  point *d_input;
  point *d_output;
  hipMalloc(&d_input, input_size * sizeof(point));
  hipMalloc(&d_output, output_size * sizeof(point));

  hipMemcpy(d_input, input.data.data(), input_size * sizeof(point),
             hipMemcpyHostToDevice);

  int blocks = (output_size + THREADS - 1) / THREADS;

  downsamplePointCloudKernel<<<blocks, THREADS>>>(d_input, d_output, input_size,
                                                  downsample_factor);
  hipDeviceSynchronize();

  pointcloud output;
  output.data.resize(output_size);

  hipMemcpy(output.data.data(), d_output, output_size * sizeof(point),
             hipMemcpyDeviceToHost);

  hipFree(d_input);
  hipFree(d_output);

  return output;
}

__global__ void
fps_kernel(const point *__restrict__ src, const int *__restrict__ ptr,
           const int *__restrict__ out_ptr, const int *__restrict__ start,
           float *__restrict__ dist, int *__restrict__ out, int dim) {

  const int tid = threadIdx.x;
  const int bid = blockIdx.x;

  const int start_idx = ptr[bid];
  const int end_idx = ptr[bid + 1];

  __shared__ float best_dist[THREADS];
  __shared__ int best_idx[THREADS];

  if (tid == 0) {
    out[out_ptr[bid]] = start_idx + start[bid];
  }

  for (int m = out_ptr[bid] + 1; m < out_ptr[bid + 1]; m++) {
    __syncthreads();
    int prev_idx = out[m - 1]; 

    float max_dist = -1.0f; 
    int max_idx = 0;

    for (int n = start_idx + tid; n < end_idx; n += THREADS) {
      float dx = src[prev_idx].x - src[n].x;
      float dy = src[prev_idx].y - src[n].y;
      float dz = src[prev_idx].z - src[n].z;
      float dist_to_prev = dx * dx + dy * dy + dz * dz;

      if (dist_to_prev < dist[n]) {
        dist[n] = dist_to_prev;
      }

      if (dist[n] > max_dist) {
        max_dist = dist[n];
        max_idx = n;
      }
    }

    best_dist[tid] = max_dist;
    best_idx[tid] = max_idx;
    __syncthreads();

    for (int offset = THREADS / 2; offset > 0; offset >>= 1) {
      if (tid < offset) {
        if (best_dist[tid + offset] > best_dist[tid]) {
          best_dist[tid] = best_dist[tid + offset];
          best_idx[tid] = best_idx[tid + offset];
        }
      }
      __syncthreads();
    }

    if (tid == 0) {
      out[m] = best_idx[0];
    }
  }
}

pointcloud farthestPointSampling(const pointcloud &input, float ratio,
                                 bool random_start) {
  int input_size = input.data.size();
  int dim = 3;        
  int batch_size = 1;
  int output_size = static_cast<int>(input_size * ratio);

  point *d_input;
  int *d_ptr;
  int *d_out_ptr;
  int *d_start;
  float *d_dist;
  int *d_out;

  hipMalloc(&d_input, input_size * sizeof(point));
  hipMalloc(&d_ptr, (batch_size + 1) * sizeof(int));
  hipMalloc(&d_out_ptr, (batch_size + 1) * sizeof(int));
  hipMalloc(&d_start, batch_size * sizeof(int));
  hipMalloc(&d_dist, input_size * sizeof(float));
  hipMalloc(&d_out, output_size * sizeof(int));

  hipMemcpy(d_input, input.data.data(), input_size * sizeof(point),
             hipMemcpyHostToDevice);

  std::vector<int> ptr = {0, input_size};
  std::vector<int> out_ptr = {0, output_size};
  hipMemcpy(d_ptr, ptr.data(), (batch_size + 1) * sizeof(int),
             hipMemcpyHostToDevice);
  hipMemcpy(d_out_ptr, out_ptr.data(), (batch_size + 1) * sizeof(int),
             hipMemcpyHostToDevice);

  std::vector<int> start(batch_size, 0);
  if (random_start) {
    for (int i = 0; i < batch_size; i++) {
      start[i] = rand() % (ptr[i + 1] - ptr[i]);
    }
  }
  hipMemcpy(d_start, start.data(), batch_size * sizeof(int),
             hipMemcpyHostToDevice);

  std::vector<float> dist(input_size,
                          1e10f); 
  hipMemcpy(d_dist, dist.data(), input_size * sizeof(float),
             hipMemcpyHostToDevice);

  fps_kernel<<<batch_size, THREADS>>>(d_input, d_ptr, d_out_ptr, d_start,
                                      d_dist, d_out, dim);
  hipDeviceSynchronize();

  std::vector<int> out_indices(output_size);
  hipMemcpy(out_indices.data(), d_out, output_size * sizeof(int),
             hipMemcpyDeviceToHost);

  pointcloud output;
  output.data.resize(output_size);
  for (int i = 0; i < output_size; i++) {
    output.data[i] = input.data[out_indices[i]];
  }

  hipFree(d_input);
  hipFree(d_ptr);
  hipFree(d_out_ptr);
  hipFree(d_start);
  hipFree(d_dist);
  hipFree(d_out);

  return output;
}

__global__ void radius_kernel(const point *__restrict__ x,
                              const point *__restrict__ y,
                              const int64_t *__restrict__ ptr_x,
                              const int64_t *__restrict__ ptr_y,
                              int64_t *__restrict__ row,
                              int64_t *__restrict__ col, const float r_squared,
                              const int n, const int m, const int num_examples,
                              const int max_num_neighbors) {

  const int n_y = blockIdx.x * blockDim.x + threadIdx.x;
  if (n_y >= m)
    return;

  int count = 0;
  const int64_t example_idx = get_example_idx(n_y, ptr_y, num_examples);

  for (int64_t n_x = ptr_x[example_idx]; n_x < ptr_x[example_idx + 1]; n_x++) {
    float dx = x[n_x].x - y[n_y].x;
    float dy = x[n_x].y - y[n_y].y;
    float dz = x[n_x].z - y[n_y].z;
    float dist = dx * dx + dy * dy + dz * dz;

    if (dist < r_squared) {
      row[n_y * max_num_neighbors + count] = n_y;
      col[n_y * max_num_neighbors + count] = n_x;
      count++;
    }

    if (count >= max_num_neighbors)
      break;
  }
}

std::pair<std::vector<int64_t>, std::vector<int64_t>>
radius(const pointcloud &x_cloud, const pointcloud &y_cloud, float radius,
       int max_num_neighbors) {
  int n = x_cloud.data.size();
  int m = y_cloud.data.size();

  std::vector<int64_t> ptr_x = {
      0, n};
  std::vector<int64_t> ptr_y = {
      0, m};

  int num_examples = ptr_x.size() - 1;

  std::vector<int64_t> row(m * max_num_neighbors, -1);
  std::vector<int64_t> col(m * max_num_neighbors, -1);

  point *d_x, *d_y;
  int64_t *d_ptr_x, *d_ptr_y, *d_row, *d_col;

  hipMalloc(&d_x, n * sizeof(point));
  hipMalloc(&d_y, m * sizeof(point));
  hipMalloc(&d_ptr_x, ptr_x.size() * sizeof(int64_t));
  hipMalloc(&d_ptr_y, ptr_y.size() * sizeof(int64_t));
  hipMalloc(&d_row, row.size() * sizeof(int64_t));
  hipMalloc(&d_col, col.size() * sizeof(int64_t));

  hipMemcpy(d_x, x_cloud.data.data(), n * sizeof(point),
             hipMemcpyHostToDevice);
  hipMemcpy(d_y, y_cloud.data.data(), m * sizeof(point),
             hipMemcpyHostToDevice);
  hipMemcpy(d_ptr_x, ptr_x.data(), ptr_x.size() * sizeof(int64_t),
             hipMemcpyHostToDevice);
  hipMemcpy(d_ptr_y, ptr_y.data(), ptr_y.size() * sizeof(int64_t),
             hipMemcpyHostToDevice);

  dim3 BLOCKS((m + THREADS - 1) / THREADS);
  radius_kernel<<<BLOCKS, THREADS>>>(d_x, d_y, d_ptr_x, d_ptr_y, d_row, d_col,
                                     radius * radius, n, m, num_examples,
                                     max_num_neighbors);
  hipDeviceSynchronize();

  hipMemcpy(row.data(), d_row, row.size() * sizeof(int64_t),
             hipMemcpyDeviceToHost);
  hipMemcpy(col.data(), d_col, col.size() * sizeof(int64_t),
             hipMemcpyDeviceToHost);

  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_ptr_x);
  hipFree(d_ptr_y);
  hipFree(d_row);
  hipFree(d_col);

  std::vector<int64_t> valid_row, valid_col;
  for (size_t i = 0; i < row.size(); i++) {
    if (row[i] != -1) {
      valid_row.push_back(row[i]);
      valid_col.push_back(col[i]);
    }
  }

  return std::make_pair(valid_row, valid_col);
}